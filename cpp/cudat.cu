#include "hip/hip_runtime.h"
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.1\include\hip/hip_runtime.h"
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.1\include\"

#include <iostream>
using namespace std;
__global__ void add(int *a, int *b, int *c) {
    int i=threadIdx.x;
    c[i]=a[i]+b[i];

}

int main() {

    int x[]={1,2,3,4,5,6,7,8,9,10};
    int y[]={1,4,9,16,25,36,49,64,81,100};
    int z[sizeof(x)]={0};
    // create device pointers
    int *d_x=0, *d_y=0, *d_z=0;
    // allocate memory on device
    hipMalloc(&d_x, sizeof(x));
    hipMalloc(&d_y, sizeof(y));
    hipMalloc(&d_z, sizeof(z));
    // copy data from host to device
    hipMemcpy(d_x, x, sizeof(x), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(y), hipMemcpyHostToDevice);
  
  add<<<1,sizeof(x)/sizeof(int)>>>(d_x,d_y,d_z);

  hipMemcpy(z, d_z, sizeof(z), hipMemcpyDeviceToHost);
  
    return 0;
}

